#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "kernel.h"
int main()
{
	float timerValueGPU, timerValueCPU;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float* hA;
	float* dA;
	float* hB;
	float* dB;
	float* hC;
	float* dC;

	int nStream = 1; //number of CUDA threads
	int  N_thread = 512;
	int size = N_thread * 50000 / nStream;  //size of each array
	int N_blocks;
	int i;
	unsigned int mem_size = sizeof(float) * size;

	hipHostMalloc((void**)&hA, mem_size * nStream);
	hipHostMalloc((void**)&hB, mem_size * nStream);
	hipHostMalloc((void**)&hC, mem_size * nStream);
	//memory allocation for arrays hA, hB, hC

	hipMalloc((void**)&dA, mem_size * nStream);
	hipMalloc((void**)&dB, mem_size * nStream);
	hipMalloc((void**)&dC, mem_size * nStream);
	//memory allocation in GPU

	for (i = 0; i < size; ++i)
	{
		hA[i] = sinf(i);
		hB[i] = cosf(2.0f*i-5.0f);
		hC[i] = 0.0f;
	}
	//filling arrays

	if ((size % N_thread) == 0) {
		N_blocks = size / N_thread;
	}
	else {
		N_blocks = (int)(size / N_thread) + 1;
	}
	dim3 blocks(N_blocks);
	
	hipStream_t stream[1];

	for (i = 0; i < nStream; ++i) {
		hipStreamCreate(&stream[i]);
	}
	//Create CUDA streams

	hipEventRecord(start, 0);

	for ( i = 0; i < nStream; ++i)
	{
		hipMemcpyAsync(dA + i * size, hA + i * size, mem_size, hipMemcpyHostToDevice, stream[i]);
		hipMemcpyAsync(dB + i * size, hB + i * size, mem_size, hipMemcpyHostToDevice, stream[i]);
	}
	//asynchronous copying from host to device
 
	for ( i = 0; i < nStream; ++i)
	{
		function << < N_blocks, N_thread, 0, stream[i] >> > (dA + i * size, dB + i * size, dC + i * size, size);
	}

	for (i = 0; i < nStream; ++i)
	{
		hipMemcpyAsync(hC + i * size, dC + i * size, mem_size, hipMemcpyDeviceToHost, stream[i]);	
	}
	//asynchronous copying from device to host

	hipDeviceSynchronize();

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timerValueGPU, start, stop);
	printf("\n GPU calculation time: %f ms\n", timerValueGPU);

	for (i = 0; i < nStream; ++i)
	{
		hipStreamDestroy(stream[i]);
	}
	// destruction of streams

	return 0;
}

